#include "hip/hip_runtime.h"
#include "rroi.h"

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "rotate_rect_ops.h"
#include "cuda_utils.h"


template <typename T>
__device__ T bilinear_interpolate(const T* bottom_data,
    const int height, const int width,
    T y, T x,
    const int index /* index for debug only*/) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    return 0;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  int y_low = (int) y;
  int x_low = (int) x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}



template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height, const int width,
    T y, T x,
    T & w1, T & w2, T & w3, T & w4,
    int & x_low, int & x_high, int & y_low, int & y_high,
    const int index /* index for debug only*/) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  y_low = (int) y;
  x_low = (int) x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = bottom_data[y_low * width + x_low];
  // T v2 = bottom_data[y_low * width + x_high];
  // T v3 = bottom_data[y_high * width + x_low];
  // T v4 = bottom_data[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}


template <typename T>
__global__ void RRoIAlignForward(const int nthreads, const T* bottom_data,
    const T spatial_scale, const int channels,
    const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int sampling_ratio,
    const T* bottom_rois, T* top_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;


    const T* offset_bottom_rois = bottom_rois + n * 6;  // batch_ind, xc, yc, w, h, angle
    int roi_batch_ind = offset_bottom_rois[0];

    // Force malformed ROIs to be 1x1
    T roi_width = max(offset_bottom_rois[3] * spatial_scale, (T)1.);
    T roi_height = max(offset_bottom_rois[4] * spatial_scale, (T)1.);

    const T* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    const T mw = 1.0 / roi_bin_grid_w;
    const T mh = 1.0 / roi_bin_grid_h;

    // compute pool points
    T P[8];
    compute_roi_pool_pts(offset_bottom_rois, P, spatial_scale, pooled_height, pooled_width, ph, pw);

    // compute line params
    T line_params[4];
    for (int i = 0; i < 2; ++i)
    {
        line_params[i * 2] = P[((i + 1) * 2) % 8] - P[i * 2];
        line_params[i * 2 + 1] = P[((i + 1) * 2) % 8 + 1] - P[i * 2 + 1];
    }

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    T output_val = 0.;
    for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
    {
      for (int ix = 0; ix < roi_bin_grid_w; ix ++)
      {
        const T x = P[0] + static_cast<T>(iy + 0.5) * line_params[0] * mh + static_cast<T>(ix + 0.5) * line_params[2] * mw;
        const T y = P[1] + static_cast<T>(iy + 0.5) * line_params[1] * mh + static_cast<T>(ix + 0.5) * line_params[3] * mw;

        T val = bilinear_interpolate(offset_bottom_data, height, width, y, x, index);
        output_val += val;
//        printf("%.2f\n", val);
      }
    }

    output_val /= count;

    top_data[index] = output_val;
  }
}


template <typename T>
__global__ void RRoIAlignBackwardFeature(const int nthreads, const T* top_diff,
    const int num_rois, const T spatial_scale,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width,
    const int sampling_ratio,
    T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    const T* offset_bottom_rois = bottom_rois + n * 6;
    int roi_batch_ind = offset_bottom_rois[0];

    // Force malformed ROIs to be 1x1
    T roi_width = max(offset_bottom_rois[3] * spatial_scale, (T)1.);
    T roi_height = max(offset_bottom_rois[4] * spatial_scale, (T)1.);

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    const T mw = 1.0 / roi_bin_grid_w;
    const T mh = 1.0 / roi_bin_grid_h;

    T* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;

    int top_offset    = (n * channels + c) * pooled_height * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    const T top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

    // compute pool points
    T P[8];
    compute_roi_pool_pts(offset_bottom_rois, P, spatial_scale, pooled_height, pooled_width, ph, pw);

    // compute line params
    T line_params[4];
    for (int i = 0; i < 2; ++i)
    {
        line_params[i * 2] = P[((i + 1) * 2) % 8] - P[i * 2];
        line_params[i * 2 + 1] = P[((i + 1) * 2) % 8 + 1] - P[i * 2 + 1];
    }

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
    {
      for (int ix = 0; ix < roi_bin_grid_w; ix ++)
      {
        const T x = P[0] + static_cast<T>(iy + 0.5) * line_params[0] * mh + static_cast<T>(ix + 0.5) * line_params[2] * mw;
        const T y = P[1] + static_cast<T>(iy + 0.5) * line_params[1] * mh + static_cast<T>(ix + 0.5) * line_params[3] * mw;

        T w1, w2, w3, w4;
        int x_low, x_high, y_low, y_high;

        bilinear_interpolate_gradient(height, width, y, x,
            w1, w2, w3, w4,
            x_low, x_high, y_low, y_high,
            index);

        T g1 = top_diff_this_bin * w1 / count;
        T g2 = top_diff_this_bin * w2 / count;
        T g3 = top_diff_this_bin * w3 / count;
        T g4 = top_diff_this_bin * w4 / count;

        if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0)
        {
          atomicAdd(offset_bottom_diff + y_low * width + x_low, static_cast<T>(g1));
          atomicAdd(offset_bottom_diff + y_low * width + x_high, static_cast<T>(g2));
          atomicAdd(offset_bottom_diff + y_high * width + x_low, static_cast<T>(g3));
          atomicAdd(offset_bottom_diff + y_high * width + x_high, static_cast<T>(g4));
        } // if
      } // ix
    } // iy
  } // CUDA_1D_KERNEL_LOOP
} // RoIAlignBackward




void vincent_rroi_align(
    int batch_size,
    int num_rois,
    int channels,
    int height,
    int width,
    int pooled_height,
    int pooled_width,
    float spatial_scale,
    float* bottom_data_d,
    float* rois_d,
    float* top_data_d,
    hipStream_t stream
    )
{
  auto top_data_size = num_rois * channels * pooled_height * pooled_width;
  dim3 grid(std::min(static_cast<long>(std::ceil(top_data_size * 1.0 / 512L)), 4096L));
  dim3 block(512);
  int sampling_ratio = 0; // default
  RRoIAlignForward<float><<<grid, block, 0, stream>>>(
      top_data_size,
      bottom_data_d,
      spatial_scale,
      channels,
      height,
      width,
      pooled_height,
      pooled_width,
      sampling_ratio, // NEW
      rois_d,
      top_data_d
      );
}

void vincent_rroi_align_backward(
    int batch_size,
    int num_rois,
    int channels,
    int height,
    int width,
    int pooled_height,
    int pooled_width,
    float spatial_scale,
    const float* top_diff_d,
    const float* rois_d,
    float* bottom_diff_d,
    hipStream_t stream
    )
{
  
  auto top_data_size = num_rois * channels * pooled_height * pooled_width;
  dim3 grid(std::min(static_cast<long>(std::ceil(top_data_size * 1.0 / 512L)), 4096L));
  dim3 block(512);
  int sampling_ratio = 0; // default
  RRoIAlignBackwardFeature<float><<<grid, block, 0, stream>>>(
         top_data_size,
         top_diff_d,
         num_rois,
         spatial_scale,
         channels,
         height,
         width,
         pooled_height,
         pooled_width,
         sampling_ratio,
         bottom_diff_d,
         rois_d
    );

}
